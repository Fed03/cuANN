#include "hip/hip_runtime.h"
#ifndef __cuANN_utils__
#define __cuANN_utils__

#include <hipblas.h>
#include <stdexcept>
#include <thrust/gather.h>
#include "commons.h"
#include "utils.h"

namespace cuANN {
	void multiplyMatrix(const float* A, const float* B, float* result, const int rowsA, const int colsA, const int colsB) {
		const float alpha = 1.0, beta = 0.0;
		
		hipblasHandle_t handle;
		if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS) {
			throw std::runtime_error("Cannot create cuBLAS handle.");
		}

//		hipblasStatus_t status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
//			rowsA, colsB, colsA,
//			&alpha,
//			A, rowsA,
//			B, colsA,
//			&beta,
//			result, rowsA
//		);

		hipblasStatus_t status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			colsB, rowsA, colsA,
			&alpha,
			B, colsB,
			A, colsA,
			&beta,
			result, colsB
		);

		hipblasDestroy(handle);

		if (status != HIPBLAS_STATUS_SUCCESS) {
			throw std::runtime_error("Cannot perform matrix multiplication.");
		}
	}

	void radixSortMatrix(const thrust::device_vector<float> &matrix, const int rows, const int cols, thrust::device_vector<unsigned> &sortedPermutationIndexes) {
		thrust::device_vector<float> dColumn(rows);
		thrust::sequence(sortedPermutationIndexes.begin(), sortedPermutationIndexes.end());

		for (int i = (cols - 1); i >= 0; i--)
		{
			thrust::gather(
				sortedPermutationIndexes.begin(), sortedPermutationIndexes.end(),
				matrix.begin() + rows * i, dColumn.begin()
			);
			thrust::stable_sort_by_key(dColumn.begin(), dColumn.end(), sortedPermutationIndexes.begin());
		}
	}

	__global__ void addVectorFromMatrix(float* matrix, const float* vector, const int rowsA, const int colsA) {
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		int row = blockIdx.y * blockDim.y + threadIdx.y;

		if (row < rowsA && col < colsA) {
			matrix[colsA * row + col] += vector[col];
		}
	}

	__global__ void divideMatrixByScalar(float* matrix, const int scalar, const int rowsA, const int colsA) {
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		int row = blockIdx.y * blockDim.y + threadIdx.y;

		if (row < rowsA && col < colsA) {
			matrix[colsA * row + col] /= scalar;
		}
	}

	__global__ void floorMatrix(float* matrix, const int rowsA, const int colsA) {
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		int row = blockIdx.y * blockDim.y + threadIdx.y;

		if (row < rowsA && col < colsA) {
			matrix[colsA * row + col] = std::floor(matrix[colsA * row + col]);
		}
	}

	__global__ void convertMatrixFromRawMajorToColumnMajor(const float* srcMatrix, float* destMatrix, const int rows, const int cols) {
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		int row = blockIdx.y * blockDim.y + threadIdx.y;
		
		if (row < rows && col < cols) {
			int rowMajorIdx = cols * row + col;
			int colMajorIdx = rows * col + row;
			destMatrix[colMajorIdx] = srcMatrix[rowMajorIdx];
		}
	}

	__global__ void copyGivenRowsFromMatrix(
		const float* srcMatrix, float* destMatrix,
		const int srcRows, const int destRows, const int cols,
		const unsigned* rowIdxs
	) {
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		int row = blockIdx.y * blockDim.y + threadIdx.y;

		if (col < cols && row < destRows) {
			destMatrix[col * destRows + row] = srcMatrix[col * srcRows + rowIdxs[row]];
		}
	}

	// concatenate 2 matrices placing the 2nd under the 1st
	__global__ void concatenateMatricesBelow(
		const float* firstMatrix, const float* secondMatrix,
		const int firstRows, const int secondRows, const int cols,
		float* destMatrix
	){
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		int row = blockIdx.y * blockDim.y + threadIdx.y;
		int totalRows = firstRows + secondRows;

		if (row < totalRows && col < cols) {
			float value = 0.0;
			int rowIdx = row % firstRows;
			if(row < firstRows) {
				value = firstMatrix[firstRows * col + rowIdx];
			}
			if(row >= firstRows) {
				value = secondMatrix[secondRows * col + rowIdx];
			}

			destMatrix[totalRows * col + row] = value;
		}
	}

	__global__ void getActualBinIdxs(
		int* binIdxsCandidates,
		const size_t* queryHashes,
		const size_t* binCodes,
		int Q, int binsNumber
	) {
		int queryId = blockIdx.x * blockDim.x + threadIdx.x;

		if(queryId < Q && binIdxsCandidates[queryId] >= 0) {
			if (queryHashes[queryId] != binCodes[binIdxsCandidates[queryId]]) {
				binIdxsCandidates[queryId] = -1;
			}
		}
	}

	__global__ void calcSquaredDistances(
		const float* A,
		const float* B,
		int cols,
		const unsigned* rowIdxsA,
		const unsigned* rowIdxsB,
		unsigned distancesNumber,
		float* result
	) {
		__shared__ float distances[BLOCK_SIZE_STRIDE_X][BLOCK_SIZE_STRIDE_Y];

		int distanceIdx = blockDim.x * blockIdx.x + threadIdx.x;
		if (distanceIdx < distancesNumber) {
			int ARowIdx = rowIdxsA[distanceIdx];
			int BRowIdx = rowIdxsB[distanceIdx];

			float distance = 0.0;
			for (int strideIdx = threadIdx.y; strideIdx < cols; strideIdx += BLOCK_SIZE_STRIDE_Y) {
				distance += powf(A[cols * ARowIdx + strideIdx] - B[cols * BRowIdx + strideIdx], 2);
			}

			distances[threadIdx.x][threadIdx.y] = distance;
		}
		__syncthreads();

		if (threadIdx.y < 4) {
			distances[threadIdx.x][threadIdx.y] += distances[threadIdx.x][threadIdx.y + 4];
		}
		__syncthreads();

		if (threadIdx.y < 2) {
			distances[threadIdx.x][threadIdx.y] += distances[threadIdx.x][threadIdx.y + 2];
		}
		__syncthreads();

		if (threadIdx.y == 0) {
			result[distanceIdx] = distances[threadIdx.x][0] + distances[threadIdx.x][1];
		}
	}

	__device__ void hashRange(const float* iteratorBegin, const float* iteratorEnd, size_t& result) {
		size_t seed = 0;
		while(iteratorBegin != iteratorEnd) {
			seed ^= static_cast<int>(*iteratorBegin) + 0x9e3779b9 + (seed << 6) + (seed >> 2);
			++iteratorBegin;
		}
		result = seed;
	}

	__global__ void hashMatrixRows(const float* matrix, const int rows, const int cols, size_t* hashes) {
		int row = blockIdx.x * blockDim.x + threadIdx.x;

		if(row < rows) {
			size_t hash;
			hashRange(matrix + cols * row, matrix + cols * (row + 1), hash);
			hashes[row] = hash;
		}
	}

}

#endif // !__cuANN_utils__
