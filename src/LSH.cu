#ifndef __cuANN_LSH__
#define __cuANN_LSH__

#include "LSH.h"
#include "utils.h"

namespace cuANN {
	LSH::LSH(int k, int L, float w, Dataset* data) {
		this->dataset = prepareDataset(data);
		index = new Index(k, L, this->dataset, w);
	}

	LSH::~LSH(){
		delete dataset;
	}

	void LSH::buildIndex() {
		this->index->buildIndex();
	}

	std::vector<QueryResult> LSH::queryIndex(Dataset* queries, int numberOfNeighbors) {
		auto colMajorQueries = prepareDataset(queries);
		return index->query(colMajorQueries, numberOfNeighbors);
	}

	Dataset* LSH::prepareDataset(Dataset* dataset) {
		int rows = dataset->N;
		int cols = dataset->d;

		thrust::device_vector<float> srcDataset(dataset->dataset, dataset->dataset + rows * cols);
		thrust::device_vector<float> destDataset(rows * cols);

		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 dimGrid((cols + dimBlock.x - 1) / dimBlock.x, (rows + dimBlock.y - 1) / dimBlock.y);
		convertMatrixFromRawMajorToColumnMajor<<<dimGrid, dimBlock >>>(
			thrust::raw_pointer_cast(srcDataset.data()),
			thrust::raw_pointer_cast(destDataset.data()),
			rows, cols
		);

		float* colMajorDataset = (float *)malloc(rows * cols * sizeof(float));
		thrust::copy(destDataset.begin(), destDataset.end(), colMajorDataset);

		return new Dataset(colMajorDataset, rows, cols, rows);
	}
}

#endif // !__cuANN_LSH__
