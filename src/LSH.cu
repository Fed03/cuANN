#ifndef __cuANN_LSH__
#define __cuANN_LSH__

#include "LSH.h"
#include "utils.h"

namespace cuANN {
	LSH::LSH(int k, int L, float w, Dataset* data) {
		this->dataset = data;
		index = new Index(k, L, this->dataset, w);
	}

	LSH::~LSH(){
		delete dataset;
	}

	void LSH::buildIndex() {
		this->index->buildIndex();
	}

	std::vector<QueryResult> LSH::queryIndex(Dataset* queries, int numberOfNeighbors) {
		return index->query(queries, numberOfNeighbors);
	}
}

#endif // !__cuANN_LSH__
