#ifndef __cuANN_HashTable__
#define __cuANN_HashTable__

#include <thrust/gather.h>
#include <thrust/count.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include "HashTable.h"
#include "QueryBinCalculator.h"
#include "utils.h"

namespace cuANN {
	HashTable::HashTable(int k, int d, float w) {
		this->k = k;
		this->d = d;
		this->w = w;
		this->N = binsNumber = 0;
		binCodes = 0;
		projectionsMatrix = offsetVector = 0;
		binSizes = binStartingIndexes = sortedMappingIdxs = 0;
	}

	HashTable::~HashTable() {
		freeMemory();
	}

	void HashTable::allocateProjectionMemory() {
		freeProjectionMemory();

		projectionsMatrix = (float *)malloc(k * d * sizeof(float));
		offsetVector = (float *)malloc(k * sizeof(float));
		if (!(projectionsMatrix && offsetVector))
		{
			throw std::runtime_error("Cannot allocate projections memory");
		}
	}

	void HashTable::allocateBinsMemory() {
		freeBinsMemory();

		binSizes = (unsigned *) malloc(binsNumber * sizeof(unsigned));
		binStartingIndexes = (unsigned *) malloc(binsNumber * sizeof(unsigned));
		sortedMappingIdxs = (unsigned *) malloc(N * sizeof(unsigned));
		binCodes = (size_t *) malloc(binsNumber * sizeof(size_t));

		if (!(binSizes && binStartingIndexes && sortedMappingIdxs && binCodes))
		{
			throw std::runtime_error("Cannot allocate bins memory");
		}
	}

	void HashTable::freeMemory() {
		freeProjectionMemory();
		freeBinsMemory();
	}

	void HashTable::freeProjectionMemory() {
		if (projectionsMatrix)
		{
			free(projectionsMatrix);
		}
		if (offsetVector)
		{
			free(offsetVector);
		}

		projectionsMatrix = offsetVector = 0;
	}

	void HashTable::freeBinsMemory() {
		if (binSizes)
		{
			free(binSizes);
		}
		if (binStartingIndexes)
		{
			free(binStartingIndexes);
		}
		if (binCodes)
		{
			free(binCodes);
		}
		if (sortedMappingIdxs)
		{
			free(sortedMappingIdxs);
		}
		binCodes = 0;
		binSizes = binStartingIndexes = sortedMappingIdxs = 0;
	}

	void HashTable::generateProjection(hiprandGenerator_t* normalGen, hiprandGenerator_t* uniformGen) {
		ThrustFloatV dProjections(k * d);
		ThrustFloatV dOffsetVector(k);


		hiprandGenerateNormal(*normalGen, thrust::raw_pointer_cast(dProjections.data()), k * d, 0, 1);
		hiprandGenerateUniform(*uniformGen, thrust::raw_pointer_cast(dOffsetVector.data()), k);

		thrust::transform(dOffsetVector.begin(), dOffsetVector.end(),
			thrust::make_constant_iterator(w), dOffsetVector.begin(),
			thrust::multiplies<float>());

		thrust::copy(dOffsetVector.begin(), dOffsetVector.end(), offsetVector);
		thrust::copy(dProjections.begin(), dProjections.end(), projectionsMatrix);
	}

	void HashTable::hashDataset(const float* dataset, const int N) {
		this->N = N;
		ThrustFloatV dProjectedMatrix(N * k);
		projectMatrix(dataset, N, dProjectedMatrix);
		calcBins(dProjectedMatrix);
	}

	ThrustQueryResult* HashTable::query(const float* queries, const int Q) {
		ThrustFloatV dProjectedQueries(Q * k);
		projectMatrix(queries, Q, dProjectedQueries);

		auto dQueriesBinIdxs = QueryBinCalculator::getBinsForProjectedQueries(
			dProjectedQueries,
			Q, k, binsNumber,
			binCodes
		);

		ThrustHIntV queriesBinIdxs(dQueriesBinIdxs);
		ThrustHUnsignedV resultIdxsForQueriesSizes(Q, 0);
		ThrustHUnsignedV resultIdxsForQueriesStartingIdxs(Q, 0);
		unsigned totalSize = 0;
		for (int query = 0; query < Q; ++query) {
			resultIdxsForQueriesStartingIdxs[query] = totalSize;
			if (queriesBinIdxs[query] != -1) {
				resultIdxsForQueriesSizes[query] = binSizes[queriesBinIdxs[query]];
				totalSize += resultIdxsForQueriesSizes[query];
			}
		}

		ThrustHUnsignedV resultIdxsForQueries(totalSize);
		for (int query = 0; query < Q; ++query) {
			if (queriesBinIdxs[query] != -1) {
				thrust::copy_n(
					sortedMappingIdxs + binStartingIndexes[queriesBinIdxs[query]],
					resultIdxsForQueriesSizes[query],
					resultIdxsForQueries.begin() + resultIdxsForQueriesStartingIdxs[query]
				);
			}
		}

		return new ThrustQueryResult(
			resultIdxsForQueriesStartingIdxs,
			resultIdxsForQueriesSizes,
			resultIdxsForQueries,
			Q, totalSize
		);
	}

	void HashTable::calcBins(const ThrustFloatV& dProjectedMatrix) {
		thrust::device_vector<size_t> hashes(N);

		dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
		dim3 dimGrid((N + dimBlock.x - 1)/dimBlock.x);
		hashMatrixRows<<<dimGrid, dimBlock>>>(
			thrust::raw_pointer_cast(dProjectedMatrix.data()),
			N, k,
			thrust::raw_pointer_cast(hashes.data())
		);

		ThrustUnsignedV dSortedPermutationIndx(N);
		thrust::sequence(dSortedPermutationIndx.begin(), dSortedPermutationIndx.end());
		thrust::stable_sort_by_key(hashes.begin(), hashes.end(), dSortedPermutationIndx.begin());

		ThrustBoolV diff(N);
		thrust::adjacent_difference(hashes.begin(), hashes.end(), diff.begin());
		thrust::transform(diff.begin(), diff.end(), diff.begin(), isDifferentFromZero());
		thrust::fill_n(diff.begin(), 1, true);

		binsNumber = thrust::count(diff.begin(), diff.end(), true);

		auto dBinStartingIndexes = computeStartingIndices(diff);
		auto dBinSizes = computeBinSizes(dBinStartingIndexes);
		auto dBinCodes = extractBinsCode(hashes, dBinStartingIndexes);

		allocateBinsMemory();

		thrust::copy(dSortedPermutationIndx.begin(), dSortedPermutationIndx.end(), sortedMappingIdxs);
		thrust::copy(dBinStartingIndexes.begin(), dBinStartingIndexes.end(), binStartingIndexes);
		thrust::copy(dBinSizes.begin(), dBinSizes.end(), binSizes);
		thrust::copy(dBinCodes.begin(), dBinCodes.end(), binCodes);
	}

	thrust::device_vector<size_t> HashTable::extractBinsCode(
		const thrust::device_vector<size_t>& hashes,
		const ThrustUnsignedV& startingIndices
	) {
		thrust::device_vector<size_t> hashCodes(binsNumber);

		thrust::gather(startingIndices.begin(), startingIndices.end(), hashes.begin(), hashCodes.begin());

		return hashCodes;
	}

	ThrustUnsignedV HashTable::originalDatasetIdxsFromStartingIdxs(
		const ThrustUnsignedV& startingIndices,
		const ThrustUnsignedV& dSortedPermutationIndx
	) {
		ThrustUnsignedV originalIdxs(startingIndices.size());

		thrust::gather(startingIndices.begin(), startingIndices.end(), dSortedPermutationIndx.begin(), originalIdxs.begin());

		return originalIdxs;
	}

	ThrustUnsignedV HashTable::computeBinSizes(const ThrustUnsignedV& startingIndices) {
		ThrustUnsignedV sizes(binsNumber);

		thrust::adjacent_difference(
			startingIndices.begin() + 1, startingIndices.end(),
			sizes.begin()
		);

		sizes[binsNumber - 1] = N - startingIndices.back();

		return sizes;
	}

	ThrustUnsignedV HashTable::computeStartingIndices(const ThrustBoolV& diff) {
		ThrustUnsignedV startingIndices(binsNumber, 0);

		ThrustUnsignedV mapping(N);
		thrust::sequence(mapping.begin(), mapping.end());

		thrust::copy_if(mapping.begin(), mapping.end(), diff.begin(), startingIndices.begin(), isTrue());

		return startingIndices;
	}

	ThrustBoolV HashTable::areRowsDifferentFromTheOneAbove(const ThrustFloatV& matrix, const ThrustUnsignedV& dSortedPermutationIndx){
		ThrustBoolV rowsDiff(N);
		ThrustFloatV dColumn(N);

		thrust::fill(rowsDiff.begin(), rowsDiff.end(), false);

		for (int col = 0; col < k; col++)
		{
			thrust::gather(
				dSortedPermutationIndx.begin(), dSortedPermutationIndx.end(),
				matrix.begin() + N * col, dColumn.begin()
			);
			thrust::adjacent_difference(dColumn.begin(), dColumn.end(), dColumn.begin());
			thrust::transform(
				dColumn.begin(), dColumn.end(), rowsDiff.begin(),
				rowsDiff.begin(), getOrDefault()
			);
		}

		// just in case first row is a zero vector
		thrust::fill_n(rowsDiff.begin(), 1, true);

		return rowsDiff;
	}

	void HashTable::projectMatrix(const float* dataset, const int N, ThrustFloatV& dProjectedMatrix) {
		ThrustFloatV dDataset(dataset, dataset + N * d);
		ThrustFloatV dProjectionsMatrix(projectionsMatrix, projectionsMatrix + d * k);
		ThrustFloatV dOffsetVector(offsetVector, offsetVector + k);

		float * dProjectedMatrixPTR = thrust::raw_pointer_cast(dProjectedMatrix.data());

		multiplyMatrix(
			thrust::raw_pointer_cast(dDataset.data()),
			thrust::raw_pointer_cast(dProjectionsMatrix.data()),
			dProjectedMatrixPTR,
			N, d, k
		);

		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 dimGrid((k + dimBlock.x - 1)/dimBlock.x, (N + dimBlock.y - 1)/dimBlock.y);
		addVectorFromMatrix <<< dimGrid, dimBlock >>> (
			dProjectedMatrixPTR,
			thrust::raw_pointer_cast(dOffsetVector.data()),
			N, k
		);
		divideMatrixByScalar <<< dimGrid, dimBlock >>> (dProjectedMatrixPTR, w, N, k);
		floorMatrix <<< dimGrid, dimBlock >>> (dProjectedMatrixPTR, N, k);
	}
}

#endif // !__cuANN_HashTable__
