#ifndef __cuANN_HashTable__
#define __cuANN_HashTable__

#include <thrust/gather.h>
#include "HashTable.h"
#include "QueryBinCalculator.h"
#include "utils.h"

namespace cuANN {
	HashTable::HashTable(int k, int d, float w) {
		this->k = k;
		this->d = d;
		this->w = w;
		this->N = binsNumber = 0;
		projectionsMatrix = offsetVector = binCodes = 0;
		binSizes = binStartingIndexes = sortedMappingIdxs = 0;
	}

	HashTable::~HashTable() {
		freeMemory();
	}

	void HashTable::allocateProjectionMemory() {
		freeProjectionMemory();

		projectionsMatrix = (float *)malloc(k * d * sizeof(float));
		offsetVector = (float *)malloc(k * sizeof(float));
		if (!(projectionsMatrix && offsetVector))
		{
			throw std::runtime_error("Cannot allocate projections memory");
		}
	}

	void HashTable::allocateBinsMemory() {
		freeBinsMemory();

		binSizes = (unsigned *) malloc(binsNumber * sizeof(unsigned));
		binStartingIndexes = (unsigned *) malloc(binsNumber * sizeof(unsigned));
		sortedMappingIdxs = (unsigned *) malloc(N * sizeof(unsigned));
		binCodes = (float *) malloc(binsNumber * k * sizeof(float));

		if (!(binSizes && binStartingIndexes && sortedMappingIdxs && binCodes))
		{
			throw std::runtime_error("Cannot allocate bins memory");
		}
	}

	void HashTable::freeMemory() {
		freeProjectionMemory();
		freeBinsMemory();
	}

	void HashTable::freeProjectionMemory() {
		if (projectionsMatrix)
		{
			free(projectionsMatrix);
		}
		if (offsetVector)
		{
			free(offsetVector);
		}

		projectionsMatrix = offsetVector = 0;
	}

	void HashTable::freeBinsMemory() {
		if (binSizes)
		{
			free(binSizes);
		}
		if (binStartingIndexes)
		{
			free(binStartingIndexes);
		}
		if (binCodes)
		{
			free(binCodes);
		}
		if (sortedMappingIdxs)
		{
			free(sortedMappingIdxs);
		}
		binCodes = 0;
		binSizes = binStartingIndexes = sortedMappingIdxs = 0;
	}

	void HashTable::generateProjection(hiprandGenerator_t* normalGen, hiprandGenerator_t* uniformGen) {
		std::cout << "allocating proj on device. params k: " << k << " d: " << d << std::endl;
		ThrustFloatV dProjections(k * d);
		ThrustFloatV dOffsetVector(k);


		hiprandGenerateNormal(*normalGen, thrust::raw_pointer_cast(dProjections.data()), k * d, 0, 1);
		hiprandGenerateUniform(*uniformGen, thrust::raw_pointer_cast(dOffsetVector.data()), k);

		thrust::transform(dOffsetVector.begin(), dOffsetVector.end(),
			thrust::make_constant_iterator(w), dOffsetVector.begin(),
			thrust::multiplies<float>());

		thrust::copy(dOffsetVector.begin(), dOffsetVector.end(), offsetVector);
		thrust::copy(dProjections.begin(), dProjections.end(), projectionsMatrix);
	}

	void HashTable::hashDataset(const float* dataset, const int N) {
		ThrustFloatV dProjectedMatrix(N * k);
		projectMatrix(dataset, N, dProjectedMatrix);
		calcBins(dProjectedMatrix);
	}

	ThrustQueryResult* HashTable::query(const float* queries, const int Q) {
		ThrustFloatV dProjectedQueries(Q * k);
		projectMatrix(queries, Q, dProjectedQueries);

		auto dQueriesBinIdxs = QueryBinCalculator::getBinsForProjectedQueries(
			dProjectedQueries,
			Q, k, binsNumber,
			binCodes
		);

		ThrustHIntV queriesBinIdxs(dQueriesBinIdxs);
		ThrustHUnsignedV resultIdxsForQueriesSizes(Q, 0);
		ThrustHUnsignedV resultIdxsForQueriesStartingIdxs(Q, 0);
		unsigned totalSize = 0;
		for (int query = 0; query < Q; ++query) {
			resultIdxsForQueriesStartingIdxs[query] = totalSize;
			if (queriesBinIdxs[query] != -1) {
				resultIdxsForQueriesSizes[query] = binSizes[queriesBinIdxs[query]];
				totalSize += resultIdxsForQueriesSizes[query];
			}
		}

		ThrustHUnsignedV resultIdxsForQueries(totalSize);
		for (int query = 0; query < Q; ++query) {
			if (queriesBinIdxs[query] != -1) {
				thrust::copy_n(
					sortedMappingIdxs + binStartingIndexes[queriesBinIdxs[query]],
					resultIdxsForQueriesSizes[query],
					resultIdxsForQueries.begin() + resultIdxsForQueriesStartingIdxs[query]
				);
			}
		}

		return new ThrustQueryResult(
			resultIdxsForQueriesStartingIdxs,
			resultIdxsForQueriesSizes,
			resultIdxsForQueries,
			Q, totalSize
		);
	}

	void HashTable::calcBins(const ThrustFloatV& dProjectedMatrix) {
		ThrustUnsignedV dSortedPermutationIndx(N);

		radixSortMatrix(dProjectedMatrix, N, k, dSortedPermutationIndx);
		auto diff = areRowsDifferentFromTheOneAbove(dProjectedMatrix, dSortedPermutationIndx);

		binsNumber = thrust::count(diff.begin(), diff.end(), true);
		auto dBinStartingIndexes = computeStartingIndices(diff);
		auto dBinSizes = computeBinSizes(dBinStartingIndexes);
		auto dBinCodes = extractBinsCode(dProjectedMatrix, dBinStartingIndexes, dSortedPermutationIndx);

		allocateBinsMemory();

		thrust::copy(dSortedPermutationIndx.begin(), dSortedPermutationIndx.end(), sortedMappingIdxs);
		thrust::copy(dBinStartingIndexes.begin(), dBinStartingIndexes.end(), binStartingIndexes);
		thrust::copy(dBinSizes.begin(), dBinSizes.end(), binSizes);
		thrust::copy(dBinCodes.begin(), dBinCodes.end(), binCodes);
	}

	ThrustFloatV HashTable::extractBinsCode(
		const ThrustFloatV& dProjectedMatrix,
		const ThrustUnsignedV& startingIndices,
		const ThrustUnsignedV& dSortedPermutationIndx
	) {
		ThrustFloatV codes(binsNumber * k);
		auto originalIdxs = originalDatasetIdxsFromStartingIdxs(startingIndices, dSortedPermutationIndx);

		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 dimGrid((k + dimBlock.x - 1)/dimBlock.x, (binsNumber + dimBlock.y - 1)/dimBlock.y);
		copyGivenRowsFromMatrix<<<dimGrid, dimBlock>>>(
			thrust::raw_pointer_cast(dProjectedMatrix.data()),
			thrust::raw_pointer_cast(codes.data()),
			N, binsNumber, k,
			thrust::raw_pointer_cast(originalIdxs.data())
		);

		return codes;
	}

	ThrustUnsignedV HashTable::originalDatasetIdxsFromStartingIdxs(
		const ThrustUnsignedV& startingIndices,
		const ThrustUnsignedV& dSortedPermutationIndx
	) {
		ThrustUnsignedV originalIdxs(startingIndices.size());

		thrust::gather(startingIndices.begin(), startingIndices.end(), dSortedPermutationIndx.begin(), originalIdxs.begin());

		return originalIdxs;
	}

	ThrustUnsignedV HashTable::computeBinSizes(const ThrustUnsignedV& startingIndices) {
		ThrustUnsignedV sizes(binsNumber);

		thrust::adjacent_difference(
			startingIndices.begin() + 1, startingIndices.end(),
			sizes.begin()
		);

		sizes[binsNumber - 1] = N - startingIndices.back();

		return sizes;
	}

	ThrustUnsignedV HashTable::computeStartingIndices(const ThrustBoolV& diff) {
		ThrustUnsignedV startingIndices(binsNumber, 0);

		ThrustUnsignedV mapping(N);
		thrust::sequence(mapping.begin(), mapping.end());

		thrust::copy_if(mapping.begin(), mapping.end(), diff.begin(), startingIndices.begin(), isTrue());

		return startingIndices;
	}

	ThrustBoolV HashTable::areRowsDifferentFromTheOneAbove(const ThrustFloatV& matrix, const ThrustUnsignedV& dSortedPermutationIndx){
		ThrustBoolV rowsDiff(N);
		ThrustFloatV dColumn(N);

		thrust::fill(rowsDiff.begin(), rowsDiff.end(), false);

		for (int col = 0; col < k; col++)
		{
			thrust::gather(
				dSortedPermutationIndx.begin(), dSortedPermutationIndx.end(),
				matrix.begin() + N * col, dColumn.begin()
			);
			thrust::adjacent_difference(dColumn.begin(), dColumn.end(), dColumn.begin());
			thrust::transform(
				dColumn.begin(), dColumn.end(), rowsDiff.begin(),
				rowsDiff.begin(), getOrDefault()
			);
		}

		// just in case first row is a zero vector
		rowsDiff[0] = true;

		return rowsDiff;
	}

	void HashTable::projectMatrix(const float* dataset, const int N, ThrustFloatV& dProjectedMatrix) {
		ThrustFloatV dDataset(dataset, dataset + N * d);
		ThrustFloatV dProjectionsMatrix(projectionsMatrix, projectionsMatrix + d * k);
		ThrustFloatV dOffsetVector(offsetVector, offsetVector + k);

		float * dProjectedMatrixPTR = thrust::raw_pointer_cast(dProjectedMatrix.data());

		multiplyMatrix(
			thrust::raw_pointer_cast(dDataset.data()),
			thrust::raw_pointer_cast(dProjectionsMatrix.data()),
			dProjectedMatrixPTR,
			N, d, k
		);

		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 dimGrid((k + dimBlock.x - 1)/dimBlock.x, (N + dimBlock.y - 1)/dimBlock.y);
		addVectorFromMatrix <<< dimGrid, dimBlock >>> (
			dProjectedMatrixPTR,
			thrust::raw_pointer_cast(dOffsetVector.data()),
			N, k
		);
		divideMatrixByScalar <<< dimGrid, dimBlock >>> (dProjectedMatrixPTR, w, N, k);
		floorMatrix <<< dimGrid, dimBlock >>> (dProjectedMatrixPTR, N, k);
	}
}

#endif // !__cuANN_HashTable__
