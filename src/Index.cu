#ifndef __cuANN_Index__
#define __cuANN_Index__

#include "commons.h"
#include "Index.h"
#include "TableQueryResult.h"

namespace cuANN {
	Index::Index(int k, int L, Dataset * data, float w) {
		this->k = 0;
		this->L = 0;
		this->dataset = 0;
		this->w = 0.0;
		this->d = 0;
		this->N = 0;

		this->tables = std::vector<HashTable*>();
		refresh(k, L, data, w);
	};

	Index::~Index() {
		freeProjectionMemory();
		tables.clear();
	}

	bool Index::refresh(int k, int L, Dataset * data, float w) {
		this->k = k;
		this->L = L;
		this->dataset = data;
		this->w = w;

		this->d = data->d;
		this->N = data->N;
		this->tables.resize(this->L);

		try
		{
			allocateProjectionMemory();
		}
		catch (const std::exception& e)
		{
			std::cerr << e.what();
			return false;
		}

		generateRandomProjections();

		return true;
	}

	bool Index::buildIndex() {
		for(const auto &table : tables) {
			table->hashDataset(dataset->dataset, dataset->N);
		}
		return true;
	}

	void Index::query(Dataset* queries, unsigned numberOfNeighbors) {
		unsigned Q = queries->N;
		std::vector<TableQueryResult*> results;
		results.resize(L);

		unsigned totalCandidatesNumber = 0;
		for (const auto& table : tables) {
			TableQueryResult* result = table->query(queries->dataset, Q);
			totalCandidatesNumber += result->resultSetSize;
			results.push_back(result);
		}

		ThrustHUnsignedV candidatesStartingIdxs(Q, 0);
		ThrustHUnsignedV candidatesSizes(Q, 0);
		ThrustHUnsignedV candidateIdxs(totalCandidatesNumber);

		unsigned queryOffset = 0;
		for (int query = 0; query < Q; ++query) {
			candidatesStartingIdxs[query] = queryOffset;
			for(const auto& tableResult: results) {
				thrust::copy_n(
					tableResult->resultSet.begin() + tableResult->resultStartingIdxs[query],
					tableResult->resultSizes[query],
					candidateIdxs.begin() + candidatesStartingIdxs[query] + candidatesSizes[query]
				);
				candidatesSizes[query] += tableResult->resultSizes[query];
			}

			auto candidatesForQueryBegin = candidateIdxs.begin() + candidatesStartingIdxs[query];
			auto candidatesForQueryEnd = candidatesForQueryBegin + candidatesSizes[query];
			thrust::sort(candidatesForQueryBegin, candidatesForQueryEnd);

			candidatesForQueryEnd = thrust::unique(candidatesForQueryBegin, candidatesForQueryEnd);
			candidatesSizes[query] = thrust::distance(candidatesForQueryBegin, candidatesForQueryEnd);
			queryOffset += candidatesSizes[query];
		}
	}

	void Index::allocateProjectionMemory() {
		for (int i = 0; i < L; i++)
		{
			auto table = new HashTable(k, d, w);
			table->allocateProjectionMemory();
			tables.push_back(std::move(table));
		}
	}

	void Index::generateRandomProjections() {
		hiprandGenerator_t uniform;
		hiprandGenerator_t normal;

		/*CURAND_CALL(hiprandCreateGenerator(&uniform, HIPRAND_RNG_PSEUDO_DEFAULT));
		CURAND_CALL(hiprandCreateGenerator(&normal, HIPRAND_RNG_PSEUDO_DEFAULT));*/
		hiprandCreateGenerator(&uniform, HIPRAND_RNG_PSEUDO_DEFAULT);
		hiprandCreateGenerator(&normal, HIPRAND_RNG_PSEUDO_DEFAULT);

		hiprandSetPseudoRandomGeneratorSeed(uniform, (unsigned long long) time(0));
		hiprandSetPseudoRandomGeneratorSeed(normal, (unsigned long long) time(0));

		for (const auto &table : tables) {
			table->generateProjection(&normal, &uniform);
		}

		hiprandDestroyGenerator(normal);
		hiprandDestroyGenerator(uniform);
	}

	void Index::freeProjectionMemory() {
		for (const auto &table : tables) {
			table->freeMemory();
		}
	}
}

#endif // !__cuANN_Index__
