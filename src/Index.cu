#include "hip/hip_runtime.h"
#ifndef __cuANN_Index__
#define __cuANN_Index__

#include "commons.h"
#include "utils.h"
#include "Index.h"

namespace cuANN {
	Index::Index(int k, int L, Dataset * data, float w) {
		this->k = 0;
		this->L = 0;
		this->dataset = 0;
		this->w = 0.0;
		this->d = 0;
		this->N = 0;

		this->tables = std::vector<HashTable*>();
		refresh(k, L, data, w);
	};

	Index::~Index() {
		freeProjectionMemory();
		tables.clear();
	}

	bool Index::refresh(int k, int L, Dataset * data, float w) {
		this->k = k;
		this->L = L;
		this->dataset = data;
		this->w = w;

		this->d = data->d;
		this->N = data->N;
		this->tables.resize(this->L);
		std::cout << "refreshing data" << std::endl;
		try
		{
			allocateProjectionMemory();
		}
		catch (const std::exception& e)
		{
			std::cerr << e.what();
			return false;
		}
		std::cout << "generating proj" << std::endl;
		generateRandomProjections();

		return true;
	}

	bool Index::buildIndex() {
		for(const auto &table : tables) {
			table->hashDataset(dataset->dataset, dataset->N);
		}
		return true;
	}

	std::vector<QueryResult> Index::query(Dataset* queries, unsigned numberOfNeighbors) {
		unsigned Q = queries->N;
		thrust::host_vector<ThrustQueryResult*> results;
		results.resize(L);

		for (const auto& table : tables) {
			results.push_back(table->query(queries->dataset, Q));
		}

		auto mergedResult = mergeQueryResults(results, Q);

		ThrustUnsignedV dCandidatesIdxs(mergedResult->resultSet);
		auto dDistances = calculateDistances(queries->dataset, Q, dCandidatesIdxs, mergedResult);
		sortDistancesAndTheirIdxs(dDistances, dCandidatesIdxs, mergedResult);

		std::vector<QueryResult> finalResult;
		for (unsigned query = 0; query < Q; ++query) {
			std::vector<unsigned> resultIdxsForQuery;
			thrust::copy_n(
				dCandidatesIdxs.begin() + mergedResult->resultStartingIdxs[query],
				std::min(numberOfNeighbors, mergedResult->resultSizes[query]),
				resultIdxsForQuery.begin()
			);
			finalResult.emplace_back(query, std::move(resultIdxsForQuery));
		}

		return finalResult;
	}

	void Index::sortDistancesAndTheirIdxs(ThrustFloatV& dDistances, ThrustUnsignedV& dCandidatesIdxs, const ThrustQueryResult* mergedResult) {
		ThrustUnsignedV dCandidatesStartingIdxs(mergedResult->resultStartingIdxs);
		ThrustUnsignedV dCandidatesSizes(mergedResult->resultSizes);

		unsigned Q = mergedResult->Q;
		for (int query = 0; query < Q; ++query) {
			thrust::sort_by_key(
				dDistances.begin() + dCandidatesStartingIdxs[query],
				dDistances.begin() + dCandidatesStartingIdxs[query] + dCandidatesSizes[query],
				dCandidatesIdxs.begin() + dCandidatesStartingIdxs[query],
				thrust::greater<float>()
			);
		}
	}

	ThrustFloatV Index::calculateDistances(const float* queries, unsigned Q, const ThrustUnsignedV& dCandidatesIdxs, const ThrustQueryResult* result) {
		unsigned distancesNumber = result->resultSetSize;
		ThrustFloatV dDistances(distancesNumber);

		ThrustUnsignedV dQueriesIdxsToCandidates(distancesNumber);
		for (int query = 0; query < Q; ++query) {
			thrust::fill_n(
				dQueriesIdxsToCandidates.begin() + result->resultStartingIdxs[query],
				result->resultSizes[query],
				query
			);
		}

		ThrustFloatV dQueries(queries, queries + Q * d);
		ThrustFloatV dDataset(dataset->dataset, dataset->dataset + N * d);

		dim3 dimBlock(BLOCK_SIZE_STRIDE_X, BLOCK_SIZE_STRIDE_Y);
		dim3 dimGrid((distancesNumber + dimBlock.x - 1)/ dimBlock.x);

		calcSquaredDistances<<<dimGrid, dimBlock>>>(
			thrust::raw_pointer_cast(dDataset.data()), N,
			thrust::raw_pointer_cast(dQueries.data()), Q,
			d,
			thrust::raw_pointer_cast(dCandidatesIdxs.data()),
			thrust::raw_pointer_cast(dQueriesIdxsToCandidates.data()),
			distancesNumber,
			thrust::raw_pointer_cast(dDistances.data())
		);

		return dDistances;
	}

	ThrustQueryResult* Index::mergeQueryResults(thrust::host_vector<ThrustQueryResult*>& results, unsigned Q) {
		unsigned maxCandidatesNumber = getMaxCandidatesNumber(results);

		ThrustHUnsignedV candidatesStartingIdxs(Q, 0);
		ThrustHUnsignedV candidatesSizes(Q, 0);
		ThrustHUnsignedV candidateIdxs(maxCandidatesNumber);

		unsigned queryOffset = 0;
		for (int query = 0; query < Q; ++query) {
			candidatesStartingIdxs[query] = queryOffset;
			for(const auto& tableResult: results) {
				thrust::copy_n(
					tableResult->resultSet.begin() + tableResult->resultStartingIdxs[query],
					tableResult->resultSizes[query],
					candidateIdxs.begin() + candidatesStartingIdxs[query] + candidatesSizes[query]
				);
				candidatesSizes[query] += tableResult->resultSizes[query];
			}

			auto candidatesForQueryBegin = candidateIdxs.begin() + candidatesStartingIdxs[query];
			auto candidatesForQueryEnd = candidatesForQueryBegin + candidatesSizes[query];
			thrust::sort(candidatesForQueryBegin, candidatesForQueryEnd);

			candidatesForQueryEnd = thrust::unique(candidatesForQueryBegin, candidatesForQueryEnd);
			candidatesSizes[query] = thrust::distance(candidatesForQueryBegin, candidatesForQueryEnd);
			queryOffset += candidatesSizes[query];
		}

		auto totalCandidatesNumber = queryOffset;

		return new ThrustQueryResult(candidatesStartingIdxs, candidatesSizes, candidateIdxs, Q, totalCandidatesNumber);
	}

	unsigned Index::getMaxCandidatesNumber(thrust::host_vector<ThrustQueryResult*>& results) {
		thrust::host_vector<unsigned> resultsSizes(L);
		thrust::transform(results.begin(), results.end(), resultsSizes.begin(), [](ThrustQueryResult* q) {
			return q->resultSetSize;
		});
		return thrust::reduce(resultsSizes.begin(), resultsSizes.end());
	}

	void Index::allocateProjectionMemory() {
		for (int i = 0; i < L; i++)
		{
			auto table = new HashTable(k, d, w);
			table->allocateProjectionMemory();
			tables.push_back(std::move(table));
		}
	}

	void Index::generateRandomProjections() {
		hiprandGenerator_t uniform;
		hiprandGenerator_t normal;

		hiprandCreateGenerator(&uniform, HIPRAND_RNG_PSEUDO_DEFAULT);
		hiprandCreateGenerator(&normal, HIPRAND_RNG_PSEUDO_DEFAULT);

		hiprandSetPseudoRandomGeneratorSeed(uniform, (unsigned long long) time(0));
		hiprandSetPseudoRandomGeneratorSeed(normal, (unsigned long long) time(0));

		for (const auto &table : tables) {
			table->generateProjection(&normal, &uniform);
		}

		hiprandDestroyGenerator(normal);
		hiprandDestroyGenerator(uniform);
	}

	void Index::freeProjectionMemory() {
		for (const auto &table : tables) {
			table->freeMemory();
		}
	}
}

#endif // !__cuANN_Index__
