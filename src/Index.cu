#include "hip/hip_runtime.h"
#ifndef __cuANN_Index__
#define __cuANN_Index__

#include <algorithm>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <time.h>
#include "commons.h"
#include "utils.h"
#include "Index.h"

namespace cuANN {
	Index::Index(int k, int L, Dataset * data, float w) {
		this->k = 0;
		this->L = 0;
		this->dataset = 0;
		this->w = 0.0;
		this->d = 0;
		this->N = 0;

		refresh(k, L, data, w);
	};

	Index::~Index() {
		freeProjectionMemory();
		tables.clear();
	}

	bool Index::refresh(int k, int L, Dataset * data, float w) {
		this->k = k;
		this->L = L;
		this->dataset = data;
		this->w = w;

		this->d = data->d;
		this->N = data->N;
		try
		{
			allocateProjectionMemory();
		}
		catch (const std::exception& e)
		{
			std::cerr << e.what();
			return false;
		}
		generateRandomProjections();

		return true;
	}

	bool Index::buildIndex() {
		for (int i = 0; i < L; i++)
		{	
			tables[i]->hashDataset(dataset->dataset, dataset->N);
		}
		return true;
	}

	std::vector<QueryResult> Index::query(Dataset* queries, unsigned numberOfNeighbors) {
		unsigned Q = queries->N;
		thrust::host_vector<ThrustQueryResult*> results;
		results.resize(L);

		for (const auto& table : tables) {
			results.push_back(table->query(queries->dataset, Q));
		}

		auto mergedResult = mergeQueryResults(results, Q);

		ThrustUnsignedV dCandidatesIdxs(mergedResult->resultSet);
		auto dDistances = calculateDistances(queries->dataset, Q, dCandidatesIdxs, mergedResult);
		sortDistancesAndTheirIdxs(dDistances, dCandidatesIdxs, mergedResult);

		std::vector<QueryResult> finalResult;
		unsigned size;
		for (unsigned query = 0; query < Q; ++query) {
			size = std::min(numberOfNeighbors, mergedResult->resultSizes[query]);
			std::vector<unsigned> resultIdxsForQuery(size);
			thrust::copy_n(
				dCandidatesIdxs.begin() + mergedResult->resultStartingIdxs[query],
				size,
				resultIdxsForQuery.begin()
			);
			finalResult.emplace_back(query, std::move(resultIdxsForQuery), size);
		}

		return finalResult;
	}

	void Index::sortDistancesAndTheirIdxs(ThrustFloatV& dDistances, ThrustUnsignedV& dCandidatesIdxs, const ThrustQueryResult* mergedResult) {
		ThrustUnsignedV dCandidatesStartingIdxs(mergedResult->resultStartingIdxs);
		ThrustUnsignedV dCandidatesSizes(mergedResult->resultSizes);

		unsigned Q = mergedResult->Q;
		for (int query = 0; query < Q; ++query) {
			thrust::sort_by_key(
				dDistances.begin() + dCandidatesStartingIdxs[query],
				dDistances.begin() + dCandidatesStartingIdxs[query] + dCandidatesSizes[query],
				dCandidatesIdxs.begin() + dCandidatesStartingIdxs[query],
				thrust::less<float>()
			);
		}
	}

	ThrustFloatV Index::calculateDistances(const float* queries, unsigned Q, const ThrustUnsignedV& dCandidatesIdxs, const ThrustQueryResult* result) {
		unsigned distancesNumber = result->resultSetSize;
		ThrustFloatV dDistances(distancesNumber);

		ThrustUnsignedV dQueriesIdxsToCandidates(distancesNumber);
		for (int query = 0; query < Q; ++query) {
			thrust::fill_n(
				dQueriesIdxsToCandidates.begin() + result->resultStartingIdxs[query],
				result->resultSizes[query],
				query
			);
		}

		ThrustFloatV dQueries(queries, queries + Q * d);
		ThrustFloatV dDataset(dataset->dataset, dataset->dataset + N * d);

		dim3 dimBlock(BLOCK_SIZE_STRIDE_X, BLOCK_SIZE_STRIDE_Y);
		dim3 dimGrid((distancesNumber + dimBlock.x - 1)/ dimBlock.x);

		calcSquaredDistances<<<dimGrid, dimBlock>>>(
			thrust::raw_pointer_cast(dDataset.data()), N,
			thrust::raw_pointer_cast(dQueries.data()), Q,
			d,
			thrust::raw_pointer_cast(dCandidatesIdxs.data()),
			thrust::raw_pointer_cast(dQueriesIdxsToCandidates.data()),
			distancesNumber,
			thrust::raw_pointer_cast(dDistances.data())
		);

		return dDistances;
	}

	ThrustQueryResult* Index::mergeQueryResults(thrust::host_vector<ThrustQueryResult*>& results, unsigned Q) {
		unsigned maxCandidatesNumber = getMaxCandidatesNumber(results);

		ThrustHUnsignedV candidatesStartingIdxs(Q, 0);
		ThrustHUnsignedV candidatesSizes(Q, 0);
		ThrustHUnsignedV candidateIdxs(maxCandidatesNumber);

		unsigned queryOffset = 0;
		for (int query = 0; query < Q; ++query) {
			candidatesStartingIdxs[query] = queryOffset;
			for(const auto& tableResult: results) {
				thrust::copy_n(
					tableResult->resultSet.begin() + tableResult->resultStartingIdxs[query],
					tableResult->resultSizes[query],
					candidateIdxs.begin() + candidatesStartingIdxs[query] + candidatesSizes[query]
				);
				candidatesSizes[query] += tableResult->resultSizes[query];
			}

			auto candidatesForQueryBegin = candidateIdxs.begin() + candidatesStartingIdxs[query];
			auto candidatesForQueryEnd = candidatesForQueryBegin + candidatesSizes[query];
			thrust::sort(candidatesForQueryBegin, candidatesForQueryEnd);

			candidatesForQueryEnd = thrust::unique(candidatesForQueryBegin, candidatesForQueryEnd);
			candidatesSizes[query] = thrust::distance(candidatesForQueryBegin, candidatesForQueryEnd);
			queryOffset += candidatesSizes[query];
		}

		auto totalCandidatesNumber = queryOffset;
		candidateIdxs.resize(totalCandidatesNumber);

		return new ThrustQueryResult(candidatesStartingIdxs, candidatesSizes, candidateIdxs, Q, totalCandidatesNumber);
	}

	unsigned Index::getMaxCandidatesNumber(thrust::host_vector<ThrustQueryResult*>& results) {
		thrust::host_vector<unsigned> resultsSizes(L);
		thrust::transform(results.begin(), results.end(), resultsSizes.begin(), [](ThrustQueryResult* q) {
			return q->resultSetSize;
		});
		return thrust::reduce(resultsSizes.begin(), resultsSizes.end());
	}

	void Index::allocateProjectionMemory() {
		for (int i = 0; i < L; i++)
		{
			auto table = new HashTable(k, d, w);
			table->allocateProjectionMemory();
			tables.push_back(std::move(table));
		}
	}

	void Index::generateRandomProjections() {
		hiprandGenerator_t uniform;
		hiprandGenerator_t normal;

		hiprandCreateGenerator(&uniform, HIPRAND_RNG_PSEUDO_DEFAULT);
		hiprandCreateGenerator(&normal, HIPRAND_RNG_PSEUDO_DEFAULT);

		hiprandSetPseudoRandomGeneratorSeed(uniform, (unsigned long long) time(0));
		hiprandSetPseudoRandomGeneratorSeed(normal, (unsigned long long) time(0));

		for (int i = 0; i < L; i++)
		{
			tables[i]->generateProjection(&normal, &uniform);
		}

		hiprandDestroyGenerator(normal);
		hiprandDestroyGenerator(uniform);
	}

	void Index::freeProjectionMemory() {
		for (int i = 0; i < L; i++)
		{
			tables[i]->freeMemory();
		}
	}
}

#endif // !__cuANN_Index__
